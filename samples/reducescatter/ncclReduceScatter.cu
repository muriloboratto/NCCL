/*%****************************************************************************80
%  Code: 
%   ncclReduceScatter.cu
%
%  Purpose:
%   Implements a simple collective operation REDUCESCATTER using NCCL (ncclReduceScatter).
%
%  Modified:
%   Aug 18 2020 10:57 
%
%  Author:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  How to Compile:
%   nvcc ncclReduceScatter.cu -o ncclReduceScatter -lnccl 
%
%  How to Execute: 
%   ./ncclReduceScatter     
%   
%  Comments:
%
%  1) For ncclReduceScatter, in place operations are done when the per-rank pointer is located at the rank offset 
%     of the global buffer. More precisely, these calls are considered in place:
%
%         ncclReduceScatter(data, data+rank*recvcount, recvcount, datatype, op, comm, stream);
%
%  2) Simple Testbed with size problem = 4 on environment with 4 GPUs. 
%****************************************************************************80*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

__global__ void Dev_print(float *x) {
   
   int i = threadIdx.x;
  
   printf("%1.2f\t", x[i]); 
  

}/*Dev_print*/   


void print_vector(float *in, int n){

 for(int i=0; i < n; i++)
  printf("%1.2f\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]){

 /*Variables*/
  int size      = 4;
  int nGPUs     = 4;
  int recvcount = 1;
  int DeviceList[4] = {0, 1, 2, 3}; /* (GPUs Id) Testbed on environment with 4 GPUs*/
  
 /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);

  /*Allocating and initializing device buffers*/
  float** sendbuff = (float**) malloc(nGPUs * sizeof(float*));
  float** recvbuff = (float**) malloc(nGPUs * sizeof(float*));

  /*Host vectors*/ 
  float host_x0[4] = { 10,  50,  90,   130};
  float host_x1[4] = { 20,  60,  100,  140};
  float host_x2[4] = { 30,  70,  110,  150};
  float host_x3[4] = { 40,  80,  120,  160};
    
  print_vector(host_x0, size); 
  print_vector(host_x1, size);
  print_vector(host_x2, size);
  print_vector(host_x3, size);

  for (int i = 0; i < nGPUs; ++i) {

   hipSetDevice(i);

   hipMalloc(&sendbuff[i],  size * sizeof(float));
   hipMalloc(&recvbuff[i],  size * sizeof(float));

    switch(i) { /*Copy from host to devices*/
      case 0 : hipMemcpy(sendbuff[i] , host_x0,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 1 : hipMemcpy(sendbuff[i] , host_x1,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 2 : hipMemcpy(sendbuff[i] , host_x2,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 3 : hipMemcpy(sendbuff[i] , host_x3,   size * sizeof(float), hipMemcpyHostToDevice); break; 
    }

   hipStreamCreate(s+i);

  } 

  ncclGroupStart();
        
      for(int g = 0; g < nGPUs; g++) {
       hipSetDevice(g);
       ncclReduceScatter(sendbuff[g], recvbuff[g], recvcount, ncclFloat, ncclSum, comms[g], s[g]); /*All Reducing and Scattering the data on GPUs*/   
 }

  ncclGroupEnd();


  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(g); 
      printf("\nThis is device %d\n", g);
      Dev_print <<< 1, size >>> (recvbuff[g]); /*Call the CUDA Kernel: Print vector on GPUs*/
      hipDeviceSynchronize();    
  }

  printf("\n");

  for (int i = 0; i < nGPUs; ++i) { /*Synchronizing CUDA Streams*/
   hipSetDevice(i);
   hipStreamSynchronize(s[i]);
  }

  for (int i = 0; i < nGPUs; ++i) { /*Destroy CUDA Streams*/
   hipSetDevice(i);
   hipFree(sendbuff[i]);
   hipFree(recvbuff[i]);
  }

  for(int i = 0; i < nGPUs; ++i)   /*Finalizing NCCL*/
    ncclCommDestroy(comms[i]);

 /*Freeing memory*/
  hipFree(sendbuff);
  hipFree(recvbuff);

  return 0;

}/*main*/