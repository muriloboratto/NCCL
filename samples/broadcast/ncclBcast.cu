/*%****************************************************************************80
%  Code: 
%   ncclBcast.cu
%
%  Purpose:
%   Implements sample BROADCAST code using the package NCCL (ncclBcast).
%   Using 'Multiples Devices per Thread'.
%   The code multiple the vector position per 2 on GPUs.
%
%  Modified:
%   Aug 17 2020 10:57 
%
%  Author:
%    Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  How to Compile:
%   nvcc ncclBcast.cu -o ncclBcast -lnccl  
%
%  Execute: 
%   ./ncclBcast <size problem>                              
%   ./ncclBcast       8
% 
%****************************************************************************80*/


#include <hip/hip_runtime.h>
#include <nccl.h>
#include <cstdio>
#include <cstdlib>
 
__global__ void kernel(int *a) 
{
  int index = threadIdx.x;

  a[index] *= 2;
  printf("%d\t", a[index]);

}/*kernel*/
 

void print_vector(int *in, int n){

 for(int i=0; i < n; i++)
  printf("%d\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]) {

  /*Usage*/ 
  if( argc < 2 ) {
     printf("Usage:\n");
     printf("%s [size problem]\n", argv[0]);
     exit(-1);
  }

  int data_size = atoi(argv[1]) ;
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  
  int *DeviceList = (int *) malloc (nGPUs     * sizeof(int));
  int *data       = (int*)  malloc (data_size * sizeof(int));
  int **d_data    = (int**) malloc (nGPUs     * sizeof(int*));
  
  for(int i = 0; i < nGPUs; i++)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
  
  /*Population the data vector*/
  for(int i = 0; i < data_size; i++)
      data[i] = rand()%(10-2)*2;
 
  print_vector(data, data_size);
      
  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&d_data[g], data_size * sizeof(int));
     
      if(g == 0)  /*Copy from Host to Device*/
         hipMemcpy(d_data[g], data, data_size * sizeof(int), hipMemcpyHostToDevice);
  }
        
  ncclGroupStart();
 
  		for(int g = 0; g < nGPUs; g++) {
  	  	    hipSetDevice(DeviceList[g]);
    	  	    ncclBcast(d_data[g], data_size, ncclInt, 0, comms[g], s[g]); /*Broadcasting it to all*/
  		}

  ncclGroupEnd();       

  for (int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      kernel <<< 1 , data_size >>> (d_data[g]);/*Call the CUDA Kernel: The code multiple the vector position per 2 on GPUs*/
      hipDeviceSynchronize();             
  }

  printf("\n");

  for (int g = 0; g < nGPUs; g++) { /*Synchronizing CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamSynchronize(s[g]);
  }
 
  for(int g = 0; g < nGPUs; g++) {  /*Destroy CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamDestroy(s[g]);
  }

  for(int g = 0; g < nGPUs; g++)    /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(s);
  free(data); 
  free(DeviceList);

  hipFree(d_data);

  return 0;

}/*main*/

