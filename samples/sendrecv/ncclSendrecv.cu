/*%****************************************************************************80
%  Code: 
%   ncclSendRecv.cu
%
%  Purpose:
%   Implements sample send/recv code using the package NCCL (p2p).
%
%  Modified:
%   Aug 18 2020 10:57 
%
%  Author:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  How to Compile:
%   nvcc ncclSendrecv.cu -o object -lnccl  
%
%  Execute: 
%   ./object <size problem>
%   ./object      8                      
%****************************************************************************80*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <nccl.h>

__global__ void kernel(int *a) { printf("%d\t", a[threadIdx.x]); }
 
void show_all(int *in, int n){

 printf("\n");

 for(int i=0; i < n; i++)
  printf("%d\t", in[i]);

 printf("\n");

}/*show_all*/


int main(int argc, char* argv[]) {

  int size = atoi(argv[1]);

  /*Usage*/ 
  if( argc < 2 ) {
     printf("Usage:\n");
     printf("%s [size problem]\n", argv[0]);
     exit(-1);
  }
  
  /*Get current amounts number of GPU*/
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  printf("nGPUs = %d\n",nGPUs);

  /*List GPU Device*/
  int *DeviceList = (int *) malloc ( nGPUs * sizeof(int));

  for(int i = 0; i < nGPUs; ++i)
      DeviceList[i] = i;
  
  /*NCCL Init*/
  ncclComm_t* comms         = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s           = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList); 

  /*General variables*/
  int *host       = (int*) malloc(size      * sizeof(int));
  int **sendbuff  = (int**)malloc(nGPUs     * sizeof(int*));
  int **recvbuff  = (int**)malloc(nGPUs     * sizeof(int*));
  
  /*Population of vector*/
  for(int i = 0; i < size; i++)
      host[i] = i;

  show_all(host, size);

  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&sendbuff[g], size * sizeof(int));
      hipMalloc(&recvbuff[g], size * sizeof(int));
     
      if(g == 0)
        hipMemcpy(sendbuff[g], host, size * sizeof(int),hipMemcpyHostToDevice);
       
  }/*for*/
  
  /*NCCL*/
  ncclGroupStart();        
  
  	for (int g = 0; g < nGPUs; g++) {
            ncclSend(sendbuff[0], size, ncclInt, g, comms[g], s[g]);
    	    ncclRecv(recvbuff[g], size, ncclInt, g, comms[g], s[g]);
        }
  
  ncclGroupEnd();          

  for (int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      kernel <<< 1 , size >>> (sendbuff[g]); 
      kernel <<< 1 , size >>> (recvbuff[g]); 
      hipDeviceSynchronize();
  }

 printf("\n");

  for (int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamSynchronize(s[g]);
  }

  
  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamDestroy(s[g]);
  }

  for(int g = 0; g < nGPUs; g++) {
     ncclCommDestroy(comms[g]);
  }
  
  free(s);
  free(host);
  
  hipFree(sendbuff);
  hipFree(recvbuff);

  return 0;

}/*main*/