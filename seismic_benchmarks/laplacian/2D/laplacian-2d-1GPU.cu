/*%****************************************************************************80
!  Code: 
!   laplacian-2d-1GPU.cu
!
!  Purpose:
!   Implements sample 2D Laplacian Method in C/C++ code using CUDA.
!
!  Modified:
!   Aug 18 2020 10:57 
!
!  Author:
!   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
!   Reynam da Cruz Pestana <reynam 'at' ufba.br>
!
!  How to Compile:
!   nvcc laplacian-2d-1GPU.cu -o object
!
!  Execute: 
!   ./object
!
!  Comments: 
!   1) Simple Testbed with size problem n = m = 8.                                            
!     
!****************************************************************************80*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

__global__ void kernel(double *a, double *c,  int m, int n, int jsta2, int jend2, int dx, int dz) {

  int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y; 
  
  double sx, sz;
  
  if( j >= (jsta2 - 1) && j < jend2 && i >= 1 && i < (m - 1)){
      sx = a[(i-1) + j*n] + a[(i+1)+ j*n] + 2 * a[i + j*n];
      sz = a[ i + (j-1)*n] + a[i + (j+1)*n] + 2 * a[i + j*n];
      c[i + j * n] = (sx/(dx*dx)) + (sz/(dz*dz));
  }

}/*kernel*/

void show_matrix(double *a, int n){

   int i, j;

   for(i = 0; i < n; i++){
      for(j = 0; j < n; j++){
        printf("%1.2f\t", a[i + j*n]);
      }
    printf("\n");
   }

   printf("\n");

}/*show_matrix*/

void show_vector(int *a, int n){

   int i;

   for(i = 0; i < n; i++)
     printf("%d\t", a[i]);

   printf("\n\n");

}/*show_vector*/

void PARA_RANGE_1(int n1,int n2, int nprocs, int myid, int jsta, int jend, int *vector_return){

	int iwork1 = (n2 - n1 + 1) / nprocs;
	int iwork2 = (n2 - n1 + 1) % nprocs;

	jsta   = (myid * iwork1) + n1 + fmin((double)myid, (double)iwork2);
	jend   = jsta + iwork1 - 1;

	if (iwork2 > myid)
	 jend = jend + 1;

    vector_return[0] = jsta;
    vector_return[1] = jend;

} /*PARA_RANGE_1*/

int main(int argc, char *argv[]){

    int n = 8;
    int m = n;
    int myid;
    int nprocs;
    double dx, dz;
    double *a, *c;
    int jsta = 1, jend = 1, jsta2, jend2;
    int i, j;
    int *vector_return = (int *) calloc (2, sizeof(int));
     
    a  =  (double*) calloc (m * n, sizeof(double));
    c  =  (double*) calloc (m * n, sizeof(double));

    dx = 1; dz = 1;

	myid = 0;
    nprocs = 1;

	PARA_RANGE_1(1, n, nprocs, myid, jsta, jend, vector_return);

	jsta = vector_return[0];
	jend = vector_return[1];

    jsta2 = jsta;
	jend2 = jend;

    jsta2 = 2;         
    jend2 = n - 1;
	
    /*Population of the matrix*/
	for (i = 0; i < m; i++)
	   for(j = jsta-1; j < jend; j++)
	      a[i + j* n] = (i + j + 2) * 1.;  /*stored dates in column (major column)*/

    show_matrix(a, n);

	double *d_a;
	double *d_c;
	
    hipMalloc((void **) &d_a,  m * n * sizeof(double));
    hipMalloc((void **) &d_c,  m * n * sizeof(double));
      
	hipMemset(d_a, 0, m * n * sizeof(double));
	hipMemset(d_c, 0, m * n * sizeof(double));
	
    /*Copy Matrix 'a' from HOST to DEVICE*/
    hipMemcpy(d_a, a,  m * n * sizeof(double), hipMemcpyHostToDevice ) ;

    /*2D BLOCK and GRID definitions*/
    int  sizeblock = 4;
    int grid = (int) ceil( (double) n / (double) sizeblock );
    dim3 dimGrid( grid, grid );
    dim3 dimBlock(sizeblock, sizeblock);
       
    kernel<<< dimGrid, dimBlock >>>(d_a, d_c, m, n, jsta2, jend2, dx, dz);

    /*Copy Matrix 'd_c' from DEVICE to HOST*/ 
    hipMemcpy( c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost );
        
    show_matrix(c, n); 
       
    /*Free memories*/
    free(a);
    free(c);
     
    hipFree(d_a) ;
    hipFree(d_c) ;

    return 0;

}/*main*/
