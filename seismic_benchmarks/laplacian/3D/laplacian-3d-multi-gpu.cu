/*%****************************************************************************80
!  Code: 
!   laplacian-3d-multi-gpu.cu
!
!  Purpose:
!   Implements sample 3D Laplacian Method in C/C++ code using CUDA and NCCL on multi-GPU systems.
!
!  Modified:
!   Aug 13 2020 10:57 
!
!  Author:
!   Murilo do Carmo Boratto <murilo.boratto 'at' fieb.org.br>
!   Reynam da Cruz Pestana <reynam 'at' ufba.br>
!
!  How to Compile:
!   nvcc laplacian-3d-multi-gpu.cu -o object -lnccl -Xcompiler -fopenmp
!     
!    ◆ Print options: 
!         nvcc laplacian-3d-multi-gpu.cu -o object -lnccl -Xcompiler -fopenmp -DprintLabel
!
!  How to Execute: 
!   ./object
!
!  Comments: 
!   1) Approach for the 4 GPUs in the same node with size problem n = m = 8
!****************************************************************************80*/

#include <hip/hip_runtime.h>
#include <nccl.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>

__global__ void kernel(double *a, double *c, double *stencil, int nx, int ny, int nz, int ista2, int iend2, double dx, double dy, double dz) {

  int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y; 
  int k = blockIdx.z * blockDim.z + threadIdx.z; 

  double sx, sz, sy;
  
  if( k >= 1 && k < (ny - 1) && j >= 1 && j < (nx - 1) && i >= (ista2 - 1) &&  i < iend2 ) {
     sz = a[(i-1) +   j  *ny  +  k   *(nx*ny)]  + stencil[(i+1) +  j    *ny    +  k   *(nx*ny)] + 2 * a[i + j*ny + k*(nx*ny)];
     sx = a[ i    + (j-1)*ny  +  k   *(nx*ny)]  + a[ i    + (j+1) *ny    +  k   *(nx*ny)] + 2 * a[i + j*ny + k*(nx*ny)];
     sy = a[ i    +   j  *ny  + (k-1)*(nx*ny)]  + a[ i    +  j    *ny    + (k+1)*(nx*ny)] + 2 * a[i + j*ny + k*(nx*ny)];
     c[i + j*ny + k*(nx*ny)] = (sz/(dz*dz)) + (sx/(dx*dx)) + (sy/(dy*dy));
  }
      
}/*kernel*/



void show_matrix_3D(double *a, int n){

   for(int k = 0; k < n; k++){
     for(int j = 0; j < n; j++){
         for(int i = 0; i < n; i++){
           printf("%1.2f\t", a[i + j*n + k*(n*n)]); 
         }
       printf("\n");
     }
    printf("\n\n");
   }

}/*show_matrix_3D*/


void PARA_RANGE(int n1,int n2, int nprocs, int myid, int *vector_return){

	int iwork1 = (n2 - n1 + 1) / nprocs;
	int iwork2 = (n2 - n1 + 1) % nprocs;

	int jsta   = (myid * iwork1) + n1 + fmin((double) myid, (double) iwork2);
	int jend   = jsta + iwork1 - 1;

	if (iwork2 > myid)
	 jend = jend + 1;

    vector_return[0] = jsta;
    vector_return[1] = jend;

}/*PARA_RANGE*/


void SUPER_PARA_RANGE(int n, int nprocs, int myid, int *vector_return, double *host_a){   

   	PARA_RANGE(1, n, nprocs, myid, vector_return);

	int jsta = vector_return[0];
	int jend = vector_return[1];
        
    for(int k = 0; k < n; k++)
	   for(int j = 0; j < n; j++)
    	     for(int i = jsta - 1; i < jend; i++)    
                  host_a[i + j*n + k*(n*n)] = (i + j + 2) * 1.;  

}/*SUPER_PARA_RANGE*/


void freeMemoryApp(int *DeviceList, hipStream_t *s, ncclComm_t *comms, double **device_a, double **device_c, 
                   double **Solution_reduced_device, double **stencil, int nGPUs){

    	 for(int g = 0; g < nGPUs; g++){ 
      	 hipSetDevice(DeviceList[g]);
         hipStreamSynchronize(s[g]);
        }
    
        for(int g = 0; g < nGPUs; g++){  
         hipSetDevice(DeviceList[g]);
         hipStreamDestroy(s[g]);
        }

        for(int g = 0; g < nGPUs; g++)    
          ncclCommDestroy(comms[g]);
      
        hipFree(device_a);
        hipFree(device_c);
        hipFree(Solution_reduced_device);
        hipFree(stencil);

}/*freeMemoryApp*/


void definitionParticionNCCLSendRecv(int *partition, int n, int nGPUs){

 int *vector_return = (int *) calloc (2, sizeof(int));
 int i = 0;

 for(int myid=0; myid < nGPUs; ++myid){
   PARA_RANGE(1, n, nGPUs, myid, vector_return);
   partition[i++] = vector_return[1];
 }

 free(vector_return);   

}/*definitionParticionNCCLSendRecv*/


/******************************************** MAIN ***********************************************************************************/

int main(int argc, char *argv[]){

      int n = 8;                                  /*Size problem*/
      int nGPUs = 0;              
      hipGetDeviceCount(&nGPUs);                /*Definitions the numbers of GPUs of the System*/
      int *DeviceList = (int *) malloc ( nGPUs * sizeof(int));
    
      for(int i = 0; i < nGPUs; ++i)             /*Adding number of GPUs in a list*/
        DeviceList[i] = i;

      /*Inicializing NCCL*/
      ncclComm_t* comms = (ncclComm_t*)   malloc(nGPUs * sizeof(ncclComm_t));  
      hipStream_t* s   = (hipStream_t*) malloc(nGPUs * sizeof(hipStream_t));
      ncclCommInitAll(comms, nGPUs, DeviceList);

      /*General Variables*/
      double **device_a                 = (double**) malloc (nGPUs     * sizeof(double*));
      double **device_c                 = (double**) malloc (nGPUs     * sizeof(double*));
      double **Solution_reduced_device  = (double**) malloc (nGPUs     * sizeof(double*));
      double **stencil                  = (double**) malloc (nGPUs     * sizeof(double*));
     
      /* 3D GRID and SIZEBLOCK definitions*/
      int  sizeblock = n / 2 ;
      int  grid = (int) ceil( (double) n / (double) sizeblock );
      dim3 dimGrid( grid, grid, grid );
      dim3 dimBlock(sizeblock, sizeblock, sizeblock);
   
/**************************************************************************************************************************/


/*Step 1 - Divison of Integral Domain of the Matrix A on GPUs*/

      for(int myid = 0; myid < nGPUs; ++myid){
     
        double  *host_a    =  (double*) calloc (n * n * n, sizeof(double)); 
        int *vector_return = (int *) calloc (2, sizeof(int));
    
        SUPER_PARA_RANGE(n, nGPUs, myid, vector_return, host_a);   

        hipSetDevice(DeviceList[myid]);

        hipMalloc(&device_a[myid],  n  * n * n * sizeof(double));  
	      hipMalloc(&device_c[myid],  n  * n * n * sizeof(double));     
	      hipMalloc(&stencil[myid],   n  * n * n * sizeof(double));     
	      hipMalloc(&Solution_reduced_device[myid],  n  * n * n * sizeof(double));     
  
        hipMemcpy(device_a[myid], host_a,   n * n * n * sizeof(double), hipMemcpyHostToDevice) ;
 
        free(host_a);
        free(vector_return);
     
       }

      #ifdef printLabel  
      for(int myid = 0; myid < nGPUs; ++myid){
        double  *host_PrintStep1  =   (double*) calloc (n * n * n, sizeof(double)); 
        cudaMemcpy(host_PrintStep1, device_a[myid], n * n * n * sizeof(double), cudaMemcpyDeviceToHost);
        printf("GPU=%d ************************************************************************************\n\n", myid);
        show_matrix_3D(host_PrintStep1, n); 
        free(host_PrintStep1);
      }
      #endif

/**************************************************************************************************************************/

/* Step 2 - Start the NCCL - ncclSend / ncclRecv */

int *partition = (int *) calloc (nGPUs, sizeof(int));
definitionParticionNCCLSendRecv(partition, n, nGPUs);

ncclGroupStart(); 

       for(int myid = 0; myid < nGPUs; ++myid){ 
        //Send/Recv FROM GPU = 1 TO GPU = 0         
        ncclSend(device_a[1]+((partition[0] + 0*n) + 0*n),  n * n * n, ncclDouble, myid, comms[myid], s[myid]);
        ncclRecv(stencil[0] + ((partition[0] + 0*n) + 0*n), n * n * n, ncclDouble, myid, comms[myid], s[myid]); 
       }

    
ncclGroupEnd(); 

ncclGroupStart(); 
 
      for(int myid = 0; myid < nGPUs; ++myid){
        //Send/Recv FROM GPU = 2 TO GPU = 1     
        ncclSend(device_a[2]+((partition[1] +  0*n) + 0*n),  n * n * n, ncclDouble, myid, comms[myid], s[myid]);
        ncclRecv(stencil[1] + ((partition[1] +  0*n) +0*n), n * n * n, ncclDouble, myid, comms[myid], s[myid]);
      }

ncclGroupEnd(); 

ncclGroupStart(); 

      for(int myid = 0; myid < nGPUs; ++myid){
        //Send/Recv FROM GPU = 3 TO GPU = 2     
        ncclSend(device_a[3]+((partition[2] +  0*n) + 0*n), n * n * n, ncclDouble, myid, comms[myid], s[myid]);
        ncclRecv(stencil[2] + ((partition[2] +  0*n) + 0*n), n * n * n, ncclDouble, myid, comms[myid], s[myid]); 
      }

ncclGroupEnd(); 
   
    #ifdef printLabel   
     for(int mystencil = 0; mystencil < nGPUs-1; ++mystencil){      
       double  *host_PrintStep2  =  (double*) calloc (n * n * n, sizeof(double)); 
       cudaMemcpy(host_PrintStep2, stencil[mystencil], n * n * n * sizeof(double), cudaMemcpyDeviceToHost);
       printf("Stencil=%d ************************************************************************************\n\n", mystencil); 
       show_matrix_3D(host_PrintStep2, n); 
       free(host_PrintStep2); 
      }
    #endif

/***************************************************************************************************************************************/

/* Step 3 - Building kernel CUDA*/

ncclGroupStart(); 

    for(int myid = 0; myid < nGPUs; myid++){

        hipSetDevice(DeviceList[myid]);
        hipStreamCreate(&s[myid]);  
        
        double dx,dy,dz; 
        dx = dy = dz = 1.0;
 
        int *vector_return = (int *) calloc (2, sizeof(int));
        PARA_RANGE(1, n, nGPUs, myid, vector_return);

        int jsta = vector_return[0];
	      int jend = vector_return[1] + 1;
 
        printf("jsta=%d\tjend=%d\n", jsta, jend);
     
        if(myid == 0)
	        jsta = 2;

        if(myid == (nGPUs - 1))
	        jend = n - 1;
           
        kernel <<< dimGrid, dimBlock >>> (device_a[myid], device_c[myid], stencil[myid], n, n, n, jsta, jend, dx, dy, dz);
        
        ncclReduce(device_c[myid], Solution_reduced_device[myid], n * n * n, ncclDouble, ncclSum, 0, comms[myid], s[myid]);

        free(vector_return);
        
    }
 
ncclGroupEnd();   

       #ifdef printLabel
        double  *host_PrintStep3  =  (double*) calloc(n * n * n, sizeof(double)); 
        cudaMemcpy(host_PrintStep3, Solution_reduced_device[0], n * n * n * sizeof(double), cudaMemcpyDeviceToHost);
        printf("LAPLACIAN ************************************************************************************\n\n");   
        show_matrix_3D(host_PrintStep3, n); 
        free(host_PrintStep3);
       #endif
   
/**************************************************************************************************************************/

/* Step 4 - Free Memory (System Operation, CUDA and NCCL)*/

     freeMemoryApp(DeviceList, s, comms, device_a, device_c, Solution_reduced_device, stencil, nGPUs);

     return 0;

}/*main*/
